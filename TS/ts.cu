#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include "../common/city_cuda.h"
#include "../common/hip/hip_runtime_api.h"


int getTriangularIdx(int dim, int row, int col)
{
    return (dim * row) + col - (row * (row+1) / 2);
}

void updateTabuList(int *tabuList, int dim)
{
    for (int i = 0; i < dim*(dim+1)/2; i++)
    {
        if (tabuList[i] > 0)
            tabuList[i]--;
    }
}

int notInTabuList(int *tabu_list, int dim, int i, int j)
{
    if (tabu_list[getTriangularIdx(dim, i, j)] > 0)
        return false;
    else
        return true;
}

__device__ float twoOptSwap_dist(int *route, float *distArr, int dim, int m, int n)
{
    float newRouteDist = 0;

    /*
    - 0 < m < n < dim
    - connect (m-1) => (n) , (m) => (n+1) , (dim-1) => (0)
    - the rest remain the same
    */

    for (int i = 0; i < dim-1; ++i)
    {
        if (i < m-1 || i > n || (i >= m && i < n))
            newRouteDist += distArr[route[i] * dim + route[i+1]];
    }
    newRouteDist += distArr[route[m-1] * dim + route[n]];
    newRouteDist += distArr[route[m] * dim + route[n+1]];
    newRouteDist += distArr[route[dim-1] * dim + route[0]];
    return newRouteDist;
}

__global__ void searchChild(int *route, float *distArr, int dim, simpleChild_t *d_children)
{
    int j = blockIdx.x;
    int i = threadIdx.x;
    int idx = i * blockDim.x + j;
    d_children[idx].i = i;
    d_children[idx].j = j;
    if (i < j) {
        d_children[idx].dist = twoOptSwap_dist(route, distArr, dim, i, j);
    }
}

child_t* bestChild(child_t *parent, float *distArr, int dim, int *tabuList, child_t *bestSoFar)
{
    child_t *bestChild;
    bestChild = new child_t();

    size_t routeSize = sizeof(int)*dim;
    size_t distArrSize = sizeof(float)*dim*dim;
    size_t childrenSize = sizeof(simpleChild_t)*dim*dim;

    int *d_route;
    float *d_distArr;

    simpleChild_t *d_children, *h_children;
    h_children = new simpleChild_t[dim*dim];

    checkCudaErrors(hipMalloc((void**)&d_route, routeSize));
    checkCudaErrors(hipMalloc((void**)&d_distArr, distArrSize));
    checkCudaErrors(hipMalloc((void**)&d_children, childrenSize));

    checkCudaErrors(hipMemcpy(d_route, parent->route, routeSize, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_distArr, distArr, distArrSize, hipMemcpyHostToDevice));

    searchChild<<<dim, dim>>>(d_route, d_distArr, dim, d_children);
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(h_children, d_children, childrenSize, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_route));
    checkCudaErrors(hipFree(d_distArr));
    checkCudaErrors(hipFree(d_children));

    simpleChild_t tmpChild = h_children[1];
    for (int i = 0; i < dim; ++i)
    {
        for (int j = i+1; j < dim; ++j)
        {
            // printf("%f\n", h_children[i*dim+j].dist);
            if (h_children[i*dim+j].dist < bestSoFar->dist)
            {   // if is better than the global best, ignore the tabu list
                tmpChild = h_children[i*dim+j];
            }
            else if (notInTabuList(tabuList, dim, i, j))
            {
                if (h_children[i*dim+j].dist < tmpChild.dist)
                {
                    tmpChild = h_children[i*dim+j];
                }
            }
        }
    }

    int tabu_turns = dim * 0.1;
    tabuList[getTriangularIdx(dim, tmpChild.i, tmpChild.j)] += tabu_turns;

    bestChild->route = twoOptSwap(parent->route, dim, tmpChild.i, tmpChild.j);
    bestChild->dist = getDist(bestChild->route, distArr, dim);

    if (parent->dist > bestSoFar->dist)
    {
        free(parent);
    }

    free(h_children);

    return bestChild;
}

float TabuSearch(float *distArr, int dim)
{
    int i, no_improve;
    int *tabuList;
    float best_dist;
    child_t *tmpChild, *bestSoFar;

    // Init tabu list, a 2D triangle list
    tabuList = new int[dim*(dim+1)/2];
    for (i = 0; i < dim*(dim+1)/2; ++i)
    {
       tabuList[i] = 0;
    }

    bestSoFar = new child_t();
    bestSoFar->route = randRoute(dim);
    bestSoFar->dist = getDist(bestSoFar->route, distArr, dim);
    tmpChild = bestSoFar;

    no_improve = 0;
    while(no_improve < 100)
    {
        tmpChild = bestChild(tmpChild, distArr, dim, tabuList, bestSoFar);
        if (tmpChild->dist < bestSoFar->dist)
        {
            free(bestSoFar);
            bestSoFar = tmpChild;
            no_improve = 0;
        }
        else no_improve++;
        updateTabuList(tabuList, dim);
    }
    best_dist = bestSoFar->dist;
    free(bestSoFar);
    return best_dist;
}

int main(int argc, char const *argv[])
{
    int trials = 1;
    int i, dim, **city;
    float opt_value, prec_err, best_dist;
    float *distArr;

    clock_t start;
    clock_t end;

    int succ_times; // == times of finding optima / trials
    float run_time;
    float total_best_dist;

    FILE *f, *fw;
    if (argc == 2)
    {
        f = fopen(argv[1], "r");
    }
    else if (argc == 3)
    {
        f = fopen(argv[1], "r");
        trials = atoi(argv[2]);
    }
    else
    {
        printf("Usage: ts_gpu [data file] [trials]\n");
        return -1;
    }

    fw = fopen("result_gpu.txt", "w");
    dim = readHeader(f, fw);

    // read nodes
    if (dim == 442)
        city = readExp(f, dim);
    else
        city = readNorm(f, dim);

    distArr = genDistMatrix_cuda(city, dim);

    // get optimal value
    opt_value = getOptValue(dim);

    prec_err = opt_value*0.01;
    total_best_dist = 0;
    run_time = 0;
    succ_times = 0;

    // run serveral times to get average results
    for (i = 0; i < trials; ++i)
    {
        start = clock();
        best_dist = TabuSearch(distArr, dim);
        end = clock();

        run_time += (float)(end-start);
        total_best_dist += best_dist;
        printf("Shortest distance: %f\n", best_dist);

        if (best_dist <= opt_value+prec_err && best_dist >= opt_value-prec_err)
        {
            succ_times++;
        }
    }

    printf("\n");
    fprintf(fw, "Search Algorithm: Hill Climbing\n");
    fprintf(fw, "Trials: %d\n", trials);
    fprintf(fw, "Average Best Distance: %.2f\n", ((float)total_best_dist/(float)trials));
    fprintf(fw, "Average Run Time: %.2f\n", (float)(run_time/trials)/CLOCKS_PER_SEC);
    fprintf(fw, "Success Rate: %f\n", (float)((float)succ_times/(float)trials*(float)100));

    free(distArr);
    hipDeviceReset();

    return 0;
}