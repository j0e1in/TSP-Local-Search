#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include "../common/city_cuda.h"
#include "../common/hip/hip_runtime_api.h"

typedef struct child_
{
    int *route;
    float dist;
} child_t;

typedef struct simpleChild_
{
    int i; // twoOptSwap index
    int j; // twoOptSwap index
    float dist;
} simpleChild_t;



int* twoOptSwap(int *route, int dim, int m, int n)
{
    /** Neighbor is defined as two of cities swapped
    *  (2-Opt swap)
    **/
    int i, j;
    int *route_new;

    route_new = new int[dim];

    // Add route[0] to route[m-1] to route_new in order
    for (i = 0; i < m; ++i)
    {
        route_new[i] = route[i];
    }

    // Add route[m] to route[n] to route_new in reverse order
    for (i = m, j = n; i <= n; ++i, --j)
    {
        route_new[i] = route[j];
    }

    // Add route[n+1] to route[dim-1] to route_new in order
    for (i = n+1; i < dim; ++i)
    {
        route_new[i] = route[i];
    }

    return route_new;
}

__device__ float twoOptSwap_dist(int *route, float *distArr, int dim, int m, int n)
{
    float newRouteDist = 0;

    /*
    - 0 < m < n < dim
    - connect (m-1) => (n) , (m) => (n+1) , (dim-1) => (0)
    - the rest remain the same
    */

    for (int i = 0; i < dim-1; ++i)
    {
        if (i < m-1 || i > n || (i >= m && i < n))
            newRouteDist += distArr[route[i] * dim + route[i+1]];
    }
    newRouteDist += distArr[route[m-1] * dim + route[n]];
    newRouteDist += distArr[route[m] * dim + route[n+1]];
    newRouteDist += distArr[route[dim-1] * dim + route[0]];
    return newRouteDist;
}

__global__ void searchChild(int *route, float *distArr, int dim, simpleChild_t *d_children)
{
    int j = blockIdx.x;
    int i = threadIdx.x;
    int idx = i * blockDim.x + j;
    d_children[idx].i = i;
    d_children[idx].j = j;
    if (i < j) {
        d_children[idx].dist = twoOptSwap_dist(route, distArr, dim, i, j);
    } else {
        d_children[idx].dist = 0;
    }
}

child_t* bestChild(child_t *parent, float *distArr, int dim)
{
    child_t *bestChild;
    bestChild = new child_t();

    size_t routeSize = sizeof(int)*dim;
    size_t distArrSize = sizeof(float)*dim*dim;
    size_t childrenSize = sizeof(simpleChild_t)*dim*dim;

    int *d_route;
    float *d_distArr;

    simpleChild_t *d_children, *h_children;
    h_children = new simpleChild_t[dim*dim];

    checkCudaErrors(hipMalloc((void**)&d_route, routeSize));
    checkCudaErrors(hipMalloc((void**)&d_distArr, distArrSize));
    checkCudaErrors(hipMalloc((void**)&d_children, childrenSize));

    checkCudaErrors(hipMemcpy(d_route, parent->route, routeSize, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_distArr, distArr, distArrSize, hipMemcpyHostToDevice));

    searchChild<<<dim, dim>>>(d_route, d_distArr, dim, d_children);
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(h_children, d_children, childrenSize, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_route));
    checkCudaErrors(hipFree(d_distArr));
    checkCudaErrors(hipFree(d_children));

    simpleChild_t tmpChild = h_children[1];
    for (int i = 0; i < dim; ++i)
    {
        for (int j = 0; j < dim; ++j)
        {
            // if (i < j)
            // {
            //     printf("%f\n", h_children[i*dim+j].dist);
            // }
            if (i < j && h_children[i*dim+j].dist < tmpChild.dist)
            {
                tmpChild = h_children[i*dim+j];
            }
        }
    }
    bestChild->route = twoOptSwap(parent->route, dim , tmpChild.i, tmpChild.j);
    bestChild->dist = getDist(bestChild->route, distArr, dim);

    free(h_children);

    return bestChild;
}

float HillClimbing(float *distArr, int dim)
{
    int improved = true;
    float best_dist;
    child_t *bestSoFar, *tmpChild;

    bestSoFar = new child_t();
    bestSoFar->route = randRoute(dim);
    bestSoFar->dist = getDist(bestSoFar->route, distArr, dim);

    // ITERATE UNTIL HIT THE LOCAL OPTIMA
    while(improved)
    {
        improved = false;
        tmpChild = bestChild(bestSoFar, distArr, dim);

        if (tmpChild->dist < bestSoFar->dist)
        {
            free(bestSoFar);
            bestSoFar = tmpChild;
            improved = true;
        }
        else
        {
            free(tmpChild);
        }
    }
    best_dist = bestSoFar->dist;
    free(bestSoFar);
    return best_dist;
}

int main(int argc, char const *argv[])
{
    int trials = 1;
    int i, dim, **city;
    float opt_value, prec_err, best_dist;
    float *distArr;

    clock_t start;
    clock_t end;

    int succ_times; // == times of finding optima / trials
    float run_time;
    float total_best_dist;

    FILE *f, *fw;
    if (argc == 2)
    {
        f = fopen(argv[1], "r");
    }
    else if (argc == 3)
    {
        f = fopen(argv[1], "r");
        trials = atoi(argv[2]);
    }
    else
    {
        printf("Usage: main_cpu / main_gpu [alg] [data file] [trials]\n");
        return -1;
    }

    fw = fopen("result_gpu.txt", "w");
    dim = readHeader(f, fw);

    // read nodes
    if (dim == 442)
        city = readExp(f, dim);
    else
        city = readNorm(f, dim);

    distArr = genDistMatrix_cuda(city, dim);

    // get optimal value
    opt_value = getOptValue(dim);

    prec_err = opt_value*0.01;
    total_best_dist = 0;
    run_time = 0;
    succ_times = 0;

    // run serveral times to get average results
    for (i = 0; i < trials; ++i)
    {
        start = clock();
        best_dist = HillClimbing(distArr, dim);
        end = clock();

        run_time += (float)(end-start);
        total_best_dist += best_dist;
        printf("Shortest distance: %f\n", best_dist);

        if (best_dist <= opt_value+prec_err && best_dist >= opt_value-prec_err)
        {
            succ_times++;
        }
    }

    printf("\n");
    fprintf(fw, "Search Algorithm: Hill Climbing\n");
    fprintf(fw, "Trials: %d\n", trials);
    fprintf(fw, "Average Best Distance: %.2f\n", ((float)total_best_dist/(float)trials));
    fprintf(fw, "Average Run Time: %.2f\n", (float)(run_time/trials)/CLOCKS_PER_SEC);
    fprintf(fw, "Success Rate: %f\n", (float)((float)succ_times/(float)trials*(float)100));

    free(distArr);
    hipDeviceReset();

    return 0;
}